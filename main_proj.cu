
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

/*
Solution of the acoustic wave equation 
	d^2u/dx^2 + d^2u/dy^2 + d^2u/dz^2 = 1/c^2 dot d^2u/dt^2
in 3D using finite difference method.
	where u = u(x,y,z,t) - displacement vector (or acoustic pressure)
	and c - speed of sound

This program outputs .vtk file that can be opened in ParaView

compile it with
$nvcc main_proj.cu -o main_proj
then run with
$./main_proj
*/


typedef double my_type;


// Boundary conditions on the displacement vector
__global__ void define_u(my_type *d_u, int N)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;
    
    d_u[x+y*N+z*N*N] = 0;
}


// Velocity model initialization
__global__ void define_c(my_type *d_c, my_type c_max, int N)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;
    
    if (x < 105 && x > 95 && y < 105 && y > 95 && z < 105 && z > 95)
    {
        d_c[x+y*N+z*N*N] = c_max/2;
    }
    else
    {
        d_c[x+y*N+z*N*N] = c_max;
    }
}   


// 3D gaussian (initial conditions)
__global__ void define_f(my_type *f, int N, int x0, int y0, int z0)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    my_type a = 0.2;
    f[x+y*N+z*N*N]=exp(-a*( (x-x0)*(x-x0)+(y-y0)*(y-y0)+(z-z0)*(z-z0) ));
}


// Finite difference implementation
__global__ void calculate_u(my_type *dun, my_type *duc, my_type *dup, my_type *c, my_type *f, int N, my_type dt, my_type dh, my_type s)
{
	/*
    dun <==> d_u next
    duc <==> d_u current
    dup <==> d_u previous
	*/
	
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    double alpha = c[x+y*N+z*N*N]*c[x+y*N+z*N*N]*dt*dt/(dh*dh);
    if (x == blockDim.x*gridDim.x-1 || x == 0)
        return;
    if (y == blockDim.y*gridDim.y-1 || y == 0)
        return;
    if (z == blockDim.z*gridDim.z-1 || z == 0)
        return;
    dun[x+y*N+z*N*N]=s*dt*dt*f[x+y*N+z*N*N]+2*duc[x+y*N+z*N*N]-dup[x+y*N+z*N*N]+
                        alpha*(duc[x+1+y*N+z*N*N]-2*duc[x+y*N+z*N*N]+duc[x-1+y*N+z*N*N]+ 
                               duc[x+(y+1)*N+z*N*N]-2*duc[x+y*N+z*N*N]+duc[x+(y-1)*N+z*N*N]+ 
                               duc[x+y*N+(z+1)*N*N]-2*duc[x+y*N+z*N*N]+duc[x+y*N+(z-1)*N*N]);
    dup[x+y*N+z*N*N]=duc[x+y*N+z*N*N];
    duc[x+y*N+z*N*N]=dun[x+y*N+z*N*N];
}


// Source function
void ricker(my_type f0, my_type t0, my_type* time_arr, my_type* s_arr, int N_time)
{
    my_type arg;
    for (int i=0; i<N_time; i++)
    {
        arg = M_PI*f0*(time_arr[i]-t0);
        s_arr[i] = (2*arg*arg-1)*exp(-arg*arg);
    }
}


// Write solution result to .vtk file
void save_VTK(my_type* arr_3D, int N, char filename[], int i)
{
    //**************************************open_file**************************************
    FILE *writefile;
    writefile = fopen (filename, "w");
    if (writefile == NULL)
    {
        printf("error opening file\n");
        exit(-1);
    }
    //****************wrtie_VTK****************
    fprintf(writefile, "# vtk DataFile Version 2.0\n");
    fprintf(writefile, "acoustic_wave_equation_sulotion\n");
    fprintf(writefile, "ASCII\n");
    fprintf(writefile, "DATASET STRUCTURED_POINTS\n");
    fprintf(writefile, "DIMENSIONS %d %d %d\n", N, N, N);
    fprintf(writefile, "ORIGIN 0 0 0\n");
    fprintf(writefile, "SPACING 1 1 1\n");
    fprintf(writefile, "POINT_DATA %d\n", N*N*N);
    if (i == -1)
    {
        fprintf(writefile, "SCALARS vel_model float 1\n");
    }
    else
    {
        if (sizeof(my_type) == sizeof(double))
            fprintf(writefile, "SCALARS u_next double 1\n");
        else if (sizeof(my_type) == sizeof(float))
            fprintf(writefile, "SCALARS u_next float 1\n");
        else{
            fprintf(writefile, "ERROR");
            printf("ERROR! please choose float or double");}
    }
    fprintf(writefile, "LOOKUP_TABLE default\n");
    for (int i=0; i<N*N*N; i++){
        fprintf(writefile, "%.12lf ", arr_3D[i]);
    }
    //**************************************close_file**************************************
    if (fclose(writefile) == 0)
    {
        printf("writing done; file %d closed\n", i);
    }
}


int main()
{
    printf("initializing numerical model...\n");
    printf("***********************spatial_parameters***********************\n");
    const int N = 128;//[number of spatial elements along one axis]
    const my_type dh = 1; //[m]
    my_type c_max = 1000.0;//[m/s]
    printf("dh = %f [m]\n", dh);
    printf("x, y and z range = %.3f [m]\n", (my_type) N*dh);
    printf("N = %d [elements]\n", N);
    printf("****************************************************************\n");
    
    printf("*************************time_parameters*************************\n");
    my_type dt = (dh / c_max)*0.1;//[sec]
    my_type time_duration = 0.085; //[sec]
    int N_time = time_duration/dt;//[number of time elements]
    printf("dt = %f [sec]\n", dt);
    printf("time_duration = %.3f [sec]\n", time_duration);
    printf("N_time = %d [elements]\n", N_time);    
    printf("*****************************************************************\n");
    
    my_type * time_arr = (my_type*) malloc (sizeof(my_type)*N_time);
    my_type * s_arr = (my_type*) malloc (sizeof(my_type)*N_time);  
    printf("**************************initial_force**************************\n");
    int x0 = N/2;
    int y0 = N/2;
    int z0 = N/2;
    printf("source location: x=%f[m], y=%f[m], z=%f[m]\n", x0*dh, y0*dh, z0*dh);
    time_arr[0] = 0;
    for (int i=1; i<N_time; i++)
    {
        time_arr[i] = time_arr[i-1] + dt;
    }
    my_type f0 = 100; // [Hz] - ricker frequency
    my_type t0 = 0.01; // [sec] - ricker shift
    ricker(f0, t0, time_arr, s_arr, N_time);
    //**********************write_ricker_to_file**********************
    FILE* writefile;
    writefile = fopen ("ricker.txt", "w");
    if (writefile == NULL)
    {
        printf("error opening file\n");
        exit(-1);
    }
    for (int i=0; i<N_time; i++)
    {
        fprintf(writefile, "%f ", s_arr[i]);
    }
    if (fclose(writefile) == 0)
    {
        printf("writing ricker.txt done; file closed\n");
    }
    printf("*****************************************************************\n");
    printf("initializing done!\n");
    
    printf("Press ENTER to Continue\n");
    getchar();  
    
    my_type *d_u_next; // field in time t+1
    my_type *d_u_cur; // field in time t
    my_type *d_u_prev; // field in time t-1
    my_type *d_c; // velocity 
    my_type *d_f; // initial_force
    hipMalloc( (void**) &d_u_next, sizeof(my_type)*N*N*N );
    hipMalloc( (void**) &d_u_cur, sizeof(my_type)*N*N*N );
    hipMalloc( (void**) &d_u_prev, sizeof(my_type)*N*N*N );
    hipMalloc( (void**) &d_c, sizeof(my_type)*N*N*N );
    hipMalloc( (void**) &d_f, sizeof(my_type)*N*N*N );
    
    my_type *u = (my_type *) malloc (sizeof(my_type)*N*N*N); // field_host(to write it to file)
    
    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks(N/8, N/8, N/8);
    
    define_f<<<numBlocks, threadsPerBlock>>>(d_f, N, x0, y0, z0);
    define_u<<<numBlocks, threadsPerBlock>>>(d_u_next, N);
    define_u<<<numBlocks, threadsPerBlock>>>(d_u_cur, N);
    define_u<<<numBlocks, threadsPerBlock>>>(d_u_prev, N);
    define_c<<<numBlocks, threadsPerBlock>>>(d_c, c_max, N);
        
    char filename[30];
    for (int i=0; i<N_time; i++)
    {
        calculate_u<<<numBlocks, threadsPerBlock>>>(d_u_next, d_u_cur, d_u_prev, d_c, d_f, N, dt, dh, s_arr[i]);
        if (i % 10 == 0)
        {
            hipMemcpy(u, d_u_next, N*N*N*sizeof(my_type), hipMemcpyDeviceToHost);
            snprintf(filename, sizeof(filename), "./result/u_next%d.vtk", i);
            save_VTK(u, N, filename, i);
        }
    }
    printf("cuda calculating done!\n");
    
    //save velocity model to velocity_model.vtk
    my_type *c = (my_type *) malloc (sizeof(my_type)*N*N*N); // velocity_host(to write it to file)
    hipMemcpy(c, d_c, N*N*N*sizeof(my_type), hipMemcpyDeviceToHost);
    snprintf(filename, sizeof(filename), "velocity_model.vtk");
    save_VTK(c, N, filename, -1);
    free(c);
    
    free(u);
    free(time_arr);
    free(s_arr);
    hipFree(d_u_next);
    hipFree(d_u_cur);
    hipFree(d_u_prev);
    hipFree(d_c);
    hipFree(d_f);
    return 0;
}